#include "hip/hip_runtime.h"
#include <resamplers.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUTRY(e) do{hipError_t ecode=(e); if(ecode!=hipSuccess) {printf("%s(%d): %s()\n\tExpression evaluated as failure.\n\t%s\n\t%s\n",__FILE__,__LINE__,__FUNCTION__,#e,hipGetErrorString(ecode)); throw 1; }} while(0)
#include <stdio.h>
#include <stdlib.h>
#define countof(e) (sizeof(e)/sizeof(*(e)))

#define WARPS_PER_BLOCK  4
#define BLOCKSIZE       (32*WARPS_PER_BLOCK) // threads per block

#define restrict __restrict

#if 0
template <class T> struct vol_t {
    T * const restrict data;
    unsigned shape[3];
    unsigned strides[4];
};

struct tetrahedron {
    float T[9];
    float ori[3];
};

template <class T>
__global__
void
__launch_bounds__(BLOCKSIZE,1)
barycentric_kernel(vol_t<T> dst,cudateTextureObject_t src, const struct tetrads[5]) {
    
}

#endif

/* INTERFACE */

static void resample(TPixel * const restrict dst,const unsigned * const restrict dst_shape,const unsigned * const restrict dst_strides,
                     TPixel * const restrict src,const unsigned * const restrict src_shape,const unsigned * const restrict src_strides,
                     const float * restrict cubeverts) {
}

static int runTests(void);

extern "C" const struct resampler_api BarycentricGPU = {
    resample,
    runTests
};

/*       */
/* TESTS */
/*       */

#define ASSERT(e)  do{if(!(e)) {printf("%s(%d): %s()(\n\tExpression evaluated as false.\n\t%s\n",__FILE__,__LINE__,__FUNCTION__,#e); return 1; }}while(0)

/* simpleTransformWithTexture */

/* FIXME: TextureObject api is not supported on pre compute 3.0 devices 
          Need to convert to texture reference api
*/

__global__ void simpleTransformWithTexture_k(float *dst,hipTextureObject_t src,int w,int h,float th) {
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y+threadIdx.y;
    float u = (x/(float)w)-0.5f,
          v = (y/(float)h)-0.5f,
         tu = u*cosf(th)-v*sinf(th)+0.5f,
         tv = u*sinf(th)+v*cosf(th)+0.5f;
    dst[y*w+x]=tex2D<float>(src,tu,tv);
}

static int simpleTransformWithTexture(void) {
    const int w=256,h=256;
    try {
        float *dst;
        hipArray *a;
        hipTextureObject_t texture=0;
        { 
            hipChannelFormatDesc d=hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
            CUTRY(hipMallocArray(&a,&d,w,h));
        }
        // --> should copy source data in at this point (hipMemcpyToArray) <-- 
        {
            struct hipResourceDesc resource;
            {
                memset(&resource,0,sizeof(resource));
                resource.resType=hipResourceTypeArray;
                resource.res.array.array=a;
            }

            struct hipTextureDesc sampler;
            {
                memset(&sampler,0,sizeof(sampler));
                sampler.addressMode[0]  =hipAddressModeWrap;
                sampler.addressMode[1]  =hipAddressModeWrap;
                sampler.filterMode      =hipFilterModeLinear;
                sampler.readMode        =hipReadModeElementType; //??
                sampler.normalizedCoords=1;
            }
            CUTRY(hipCreateTextureObject(&texture,&resource,&sampler,0));
        }

        float *out;
        CUTRY(hipMalloc(&out,w*h*sizeof(float)));

        dim3 block(16,16),
             grid((w+block.x-1)/block.x,
                  (h+block.y-1)/block.y);
        simpleTransformWithTexture_k<<<grid,block>>>(out,texture,w,h,15*3.14159f/180.0f);

        // clean up
        hipFree(out);
        hipDestroyTextureObject(texture);
        hipFreeArray(a);

    } catch(int) {
        return 1;
    }
    return 0;
}

/* Test directory */

static int (*tests[])(void)={
    simpleTransformWithTexture
};

static int runTests() {
    int i;
    int nfailed=0;
    for(i=0;i<countof(tests);++i) {
        nfailed+=tests[i]();
    }
    return nfailed;
}
